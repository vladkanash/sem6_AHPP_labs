
//Func = (A + A) * B - C

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>


#define FIBER 16
#define N 2048
#define DATA_SIZE N * N * sizeof(int)

__global__ void kernel(int *a, int *x, int *b, int *r) {
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int dx = blockDim.x;
	int dy = blockDim.y;

	int i = bx * dx + tx;
	int j = by * dy + ty;
	r[i+ j * N] = (a[i + j * N] + a[i + j * N]) * b[j + i * N] - x[i + j * N];
}

using namespace std;

int* processMtrx(int* A, int* B, int* C) {
    int *R = (int*)aligned_alloc(32, DATA_SIZE);
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            R[i * N + j] = (A[i * N + j] + A[i * N + j]) * B[j * N + i] - C[i * N + j];
        }
    }
    return R;
}


void myCudaMalloc(int **ptr) {
    hipError_t error = hipMalloc((void**) ptr, DATA_SIZE);
    if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}
}

void cudaMemcpyHost2Device(int *src, int *dst) {
    hipError_t error = hipMemcpy(dst, src, DATA_SIZE, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}
}

void cudaMemcpyDevice2Host(int *src, int *dst) {
    hipError_t error = hipMemcpy(src, dst, DATA_SIZE, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}
}

bool checkForErrors(int *ptr1, int *ptr2) {
    for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
			if(ptr1[i * N + j] != ptr2[i * N + j]) {
                printf("\n%d != %d [%d]\n", ptr1[i * N + j], ptr2[i * N + j], i * N + j);
                return false;
            }

    return true;
}

int* randMtrx(int *MATR)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            MATR[i * N + j] = rand() % 1000;
        }
    }

    return MATR;
}

void print(int** r)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%u\t", r[i][j]);
            if (j == N - 1)
            {
                printf("\n");
            }
        }
    }
    printf("\n");
    printf("\n");
}

int* processCPU(int *A, int *B, int *X) {
    int* R ;

    randMtrx(A);
    randMtrx(X);
    randMtrx(B);

    struct timeval stopm, startm;
    gettimeofday(&startm, NULL);

    R = processMtrx(A, B, X);

    gettimeofday(&stopm, NULL);

    printf("runTimeCPU =  %f \n", (float)(stopm.tv_usec - startm.tv_usec) / 1000);
    return R;
}

int* processGPU(int *A, int *B, int *X) {
    int* R;
    int *Res = (int*)aligned_alloc(32, DATA_SIZE);
    memset(Res, 0, DATA_SIZE);

    myCudaMalloc(&R);
    cudaMemcpyHost2Device(Res, R);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    dim3 threads(FIBER, FIBER);
    dim3 blocks((N + (FIBER - 1)) / FIBER, (N + (FIBER - 1)) / FIBER);

    hipEventSynchronize(start);

    kernel <<< blocks, threads >>> (A, X, B, R);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("%s\n", hipGetErrorString(error));
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
	error = hipGetLastError();
    if (error != hipSuccess) {
        printf("%s\n", hipGetErrorString(error));
    }

    float timer = 0;

    hipEventElapsedTime(&timer, start, stop);
    cout << "runTimeGPU = " << timer << endl;
    hipEventRecord(start);

    cudaMemcpyDevice2Host(Res, R);
    return Res;
}

int main(int argc, char* argv[])
{
    int *dev_A, *dev_B, *dev_C;
    int *A, *B, *C;
    int *cpu_result, *gpu_result;

	A = (int*)aligned_alloc(32, DATA_SIZE);
	B = (int*)aligned_alloc(32, DATA_SIZE);
	C = (int*)aligned_alloc(32, DATA_SIZE);

    cpu_result = processCPU(A, B, C);

    myCudaMalloc(&dev_A);
    myCudaMalloc(&dev_B);
    myCudaMalloc(&dev_C);

    cudaMemcpyHost2Device(A, dev_A);
    cudaMemcpyHost2Device(B, dev_B);
    cudaMemcpyHost2Device(C, dev_C);

    gpu_result = processGPU(dev_A, dev_B, dev_C);

    if (!checkForErrors(cpu_result, gpu_result)) {
        printf("Errors occured!");
    }

}

